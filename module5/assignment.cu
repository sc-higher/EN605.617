#include "hip/hip_runtime.h"
/*
Sean Connor - February 2022
605.617 Module 4 Assignment
*/

/* ========================================================================== */

#include <stdio.h>
#include <iostream>
#include <numeric>
#include <iterator>
#include <chrono>
#include <random>

static std::random_device rd;
static std::mt19937 rng{rd()};
static std::uniform_int_distribution<int> case2_val(0,3);
static std::uniform_int_distribution<int> case3_val(0,100);

__constant__ int c_in2[4096];

/* ========================================================================== */

/**
 * @brief 
 * 
 * @param n 
 * @param a 
 * @param b 
 * @param c 
 * @return __global__ 
 */
 __global__ 
 void add(int n, int *a, int *b, int *c) {
	
	for (int i = (blockIdx.x * blockDim.x) + threadIdx.x;
		 i < n;
		 i += blockDim.x * gridDim.x)
	{
		c[i] = a[i] + b[i];
	}
	
}

// overload function for constant memory usage
__global__ 
 void add(int n, int *a, int *c) {
	
	for (int i = (blockIdx.x * blockDim.x) + threadIdx.x;
		 i < n;
		 i += blockDim.x * gridDim.x)
	{
		c[i] = a[i] + c_in2[i];
	}
	
}

// overload function for shared memory usage
__global__ 
 void add(int n, int blockSize, int *a, int *b, int *c) {
	
	extern __shared__ int s[];
		
	for (int i = (blockIdx.x * blockDim.x) + threadIdx.x;
		 i < n;
		 i += blockDim.x * gridDim.x)
	{
		s[i] = b[i];
		__syncthreads();
		c[i] = a[i] + s[i];
	}
	
}

/* ========================================================================== */

/**
 * @brief 
 * 
 * @param n 
 * @param a 
 * @param b 
 * @param c 
 * @return __global__ 
 */
__global__ 
void subtract(int n, int *a, int *b, int *c) {
	
	for (int i = (blockIdx.x * blockDim.x) + threadIdx.x;
		 i < n;
		 i += blockDim.x * gridDim.x)
	{
		c[i] = a[i] - b[i];
	}
	
}

// overload function for constant memory usage
__global__ 
void subtract(int n, int *a, int *c) {
	
	for (int i = (blockIdx.x * blockDim.x) + threadIdx.x;
		 i < n;
		 i += blockDim.x * gridDim.x)
	{
		c[i] = a[i] - c_in2[i];
	}
	
}

// overload function for shared memory usage
__global__ 
void subtract(int n, int blockSize, int *a, int *b, int *c) {
	
	extern __shared__ int s[];
		
	for (int i = (blockIdx.x * blockDim.x) + threadIdx.x;
		 i < n;
		 i += blockDim.x * gridDim.x)
	{
		s[i] = b[i];
		__syncthreads();
		c[i] = a[i] - s[i];
	}
	
}

/* ========================================================================== */

/**
 * @brief 
 * 
 * @param n 
 * @param a 
 * @param b 
 * @param c 
 * @return __global__ 
 */
__global__ 
void multiply(int n, int *a, int *b, int *c) {
	
	for (int i = (blockIdx.x * blockDim.x) + threadIdx.x;
		 i < n;
		 i += blockDim.x * gridDim.x)
	{
		c[i] = a[i] * b[i];
	}
	
}

// overload function for constant memory usage
__global__ 
void multiply(int n, int *a, int *c) {
	
	for (int i = (blockIdx.x * blockDim.x) + threadIdx.x;
		 i < n;
		 i += blockDim.x * gridDim.x)
	{
		c[i] = a[i] * c_in2[i];
	}
	
}

// overload function for shared memory usage
__global__ 
void multiply(int n, int blockSize, int *a, int *b, int *c) {
	
	extern __shared__ int s[];
		
	for (int i = (blockIdx.x * blockDim.x) + threadIdx.x;
		 i < n;
		 i += blockDim.x * gridDim.x)
	{
		s[i] = b[i];
		__syncthreads();
		c[i] = a[i] * s[i];
	}
	
}

/* ========================================================================== */

/**
 * @brief 
 * 
 * @param n 
 * @param a 
 * @param b 
 * @param c 
 * @return __global__ 
 */
__global__ 
void modulo(int n, int *a, int *b, int *c) {
	
	for (int i = (blockIdx.x * blockDim.x) + threadIdx.x;
		 i < n;
		 i += blockDim.x * gridDim.x)
	{
		c[i] = a[i] % b[i];
	}
	
}

// overload function for constant memory usage
__global__ 
void modulo(int n, int *a, int *c) {
	
	for (int i = (blockIdx.x * blockDim.x) + threadIdx.x;
		 i < n;
		 i += blockDim.x * gridDim.x)
	{
		c[i] = a[i] % c_in2[i];
	}
	
}

// overload function for shared memory usage
__global__ 
 void modulo(int n, int blockSize, int *a, int *b, int *c) {
	
	extern __shared__ int s[];
		
	for (int i = (blockIdx.x * blockDim.x) + threadIdx.x;
		 i < n;
		 i += blockDim.x * gridDim.x)
	{
		s[i] = b[i];
		__syncthreads();
		c[i] = a[i] % s[i];
	}
	
}

/* ========================================================================== */

/**
 * @brief 
 * 
 * @param array Point to array to be filled.
 * @param len Length of array to be filled.
 * @param type Type 1 = Increment 0 - len; Type 2 = 0-3 random; Type 3 = random
 */
void in_generator(int * array, int len, int type) {

	switch(type) {
		case 1:
			for (int i = 0; i < len; i++) {
				array[i] = i;
			}
			break;
		case 2:
			for (int i = 0; i < len; i++) {
				array[i] = case2_val(rng);
			}
			break;
		case 3:
			for (int i = 0; i < len; i++) {
				array[i] = case3_val(rng);
			}
			break;
		default:
			std::cout << "INVALID" << std::endl;
	}
	
}

/* ========================================================================== */

/**
 * @brief 
 * 
 * @param argc 
 * @param argv 
 */
void parse_cmdline(int argc, char **argv, int * numTh, int * blSz, int * dataSz)
{
	if (argc >= 2) {
		*numTh = atoi(argv[1]);
	}
	if (argc >= 3) {
		*blSz = atoi(argv[2]);
	}
	if (argc >= 4) {
		// *dataSz = atoi(argv[3]);
		*dataSz = 4096;
	}

	int numBlocks = *numTh / *blSz;	

	// validate command line arguments
	if (*numTh % *blSz != 0) {
		++numBlocks;
		*numTh = numBlocks * (*blSz);
		printf("Warning: Total thread count is not evenly divisible by the block size\n");
		printf("The total number of threads will be rounded up to %d\n", *numTh);
	}
	printf("================\n");
	printf("Total Threads: %d\n", *numTh);
	printf("Block Size: %d\n", *blSz);
	printf("Data Size: %d\n", *dataSz);
	printf("================\n");
}

/* ========================================================================== */

/**
 * @brief 
 * 
 * @param numBlocks 
 * @param blockSize 
 * @param dataSize 
 * @param out1 
 * @param d_in1 
 * @param d_in2 
 * @param d_out1 
 * @param type 
 */
void do_action(int numBlocks, int blockSize, int dataSize, int * d_in1, 
	int * d_in2, int * d_out1, int type) {
	
	using namespace std;

	char name[20];
	switch(type) {
		case 1:
			strcpy_s(name, 20,"add.txt");
			// add<<<numBlocks, blockSize>>>(dataSize, d_in1, d_in2, d_out1);
			break;
		case 2:
			strcpy_s(name, 20, "sub.txt");
			subtract<<<numBlocks, blockSize>>>(dataSize, d_in1, d_in2, d_out1);
			break;
		case 3:
			strcpy_s(name, 20, "mul.txt");
			multiply<<<numBlocks, blockSize>>>(dataSize, d_in1, d_in2, d_out1);
			break;
		case 4:
			strcpy_s(name, 20, "mod.txt");
			modulo<<<numBlocks, blockSize>>>(dataSize, d_in1, d_in2, d_out1);
			break;
		default:
			std::cout << "INVALID" << std::endl;
			exit(0);
	}

}

/* ========================================================================== */

/**
 * @brief 
 * 
 * @param in1 
 * @param in2 
 * @param out1 
 * @param pTotalThreads 
 * @param pBlockSize 
 * @param pDataSize 
 */
void constant_test(int * pTotalThreads, int * pBlockSize, int * pDataSize, 
	float * pTimer) {

	int size = *pDataSize * sizeof(int);
	int numBlocks = *pTotalThreads / *pBlockSize;

	// allocate host data arrays
	int *in1 = new int[*pDataSize] {0};
	int *in2 = new int[*pDataSize] {0};
	int *out1 = new int[*pDataSize] {0};
	
	// generate data and allocate device data (global)
	int *d_in1, *d_in2, *d_out1;
	in_generator(in1,*pDataSize,1);
	in_generator(in2,*pDataSize,2);
	hipMalloc((void **) &d_in1, size);
	hipMalloc((void **) &d_in2, size);
	hipMalloc((void **) &d_out1, size);
	hipMemcpy(d_in1, in1, size, hipMemcpyHostToDevice);
	hipMemcpy(d_in2, in2, size, hipMemcpyHostToDevice);
		
	// copy device global --> constant
	hipMemcpyToSymbol(HIP_SYMBOL(c_in2), d_in2, size, 0, hipMemcpyDeviceToDevice);

	// set up CUDA timing
	// https://developer.nvidia.com/blog/how-implement-performance-metrics-cuda-cc/
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// execute kernels and time
	hipEventRecord(start);	
	add<<<numBlocks, *pBlockSize>>>(*pDataSize, d_in1, d_out1);
	subtract<<<numBlocks, *pBlockSize>>>(*pDataSize, d_in1, d_out1);
	multiply<<<numBlocks, *pBlockSize>>>(*pDataSize, d_in1, d_out1);
	modulo<<<numBlocks, *pBlockSize>>>(*pDataSize, d_in1, d_out1);
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(pTimer, start, stop);	
	
	// // print statement to verify accuracy of results
	// hipMemcpy(out1, d_out1, size, hipMemcpyDeviceToHost);
	// for (int j = 0; j < *pDataSize; j+=128) {
	// 	printf("out[%d] = %d\n", j, out1[j]);
	// }
		
	// clean up
	hipFree(d_in1); hipFree(d_in2); hipFree(d_out1);
	delete [] in1; 	delete [] in2; delete [] out1;
			
}

/* ========================================================================== */

/**
 * @brief 
 * 
 * @param in1 
 * @param in2 
 * @param out1 
 * @param pTotalThreads 
 * @param pBlockSize 
 * @param pDataSize 
 */
 void shared_test(int * pTotalThreads, int * pBlockSize, int * pDataSize, 
	float * pTimer) {

	int size = *pDataSize * sizeof(int);
	int numBlocks = *pTotalThreads / *pBlockSize;

	// allocate host data arrays
	int *in1 = new int[*pDataSize] {0};
	int *in2 = new int[*pDataSize] {0};
	int *out1 = new int[*pDataSize] {0};
	
	// generate data and allocate device data (global)
	int *d_in1, *d_in2, *d_out1;
	in_generator(in1,*pDataSize,1);
	in_generator(in2,*pDataSize,2);
	hipMalloc((void **) &d_in1, size);
	hipMalloc((void **) &d_in2, size);
	hipMalloc((void **) &d_out1, size);
	hipMemcpy(d_in1, in1, size, hipMemcpyHostToDevice);
	hipMemcpy(d_in2, in2, size, hipMemcpyHostToDevice);
		
	// copy device global --> constant
	hipMemcpyToSymbol(HIP_SYMBOL(c_in2), d_in2, size, 0, hipMemcpyDeviceToDevice);

	// set up CUDA timing
	// https://developer.nvidia.com/blog/how-implement-performance-metrics-cuda-cc/
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// execute kernels and time
	int blkSzBytes = *pBlockSize*sizeof(int);
	hipEventRecord(start);	
	add<<<numBlocks, *pBlockSize, blkSzBytes>>>(*pDataSize, d_in1, d_out1);
	subtract<<<numBlocks, *pBlockSize, blkSzBytes>>>(*pDataSize, d_in1, d_out1);
	multiply<<<numBlocks, *pBlockSize, blkSzBytes>>>(*pDataSize, d_in1, d_out1);
	modulo<<<numBlocks, *pBlockSize, blkSzBytes>>>(*pDataSize, d_in1, d_out1);
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(pTimer, start, stop);	
	
	// // print statement to verify accuracy of results
	// hipMemcpy(out1, d_out1, size, hipMemcpyDeviceToHost);
	// for (int j = 0; j < *pDataSize; j+=128) {
	// 	printf("out[%d] = %d\n", j, out1[j]);
	// }
		
	// clean up
	hipFree(d_in1); hipFree(d_in2); hipFree(d_out1);
	delete [] in1; 	delete [] in2; delete [] out1;
			
}

/* ========================================================================== */

/**
 * @brief 
 * 
 * @param in1 
 * @param in2 
 * @param out1 
 * @param pTotalThreads 
 * @param pBlockSize 
 * @param pDataSize 
 */
 void global_test(int * pTotalThreads, int * pBlockSize, int * pDataSize, 
	float * pTimer) {

	int size = *pDataSize * sizeof(int);
	int numBlocks = *pTotalThreads / *pBlockSize;

	// allocate host data arrays
	int *in1 = new int[*pDataSize] {0};
	int *in2 = new int[*pDataSize] {0};
	int *out1 = new int[*pDataSize] {0};
	
	// generate data and allocate device data (global)
	int *d_in1, *d_in2, *d_out1;
	in_generator(in1,*pDataSize,1);
	in_generator(in2,*pDataSize,2);
	hipMalloc((void **) &d_in1, size);
	hipMalloc((void **) &d_in2, size);
	hipMalloc((void **) &d_out1, size);
	hipMemcpy(d_in1, in1, size, hipMemcpyHostToDevice);
	hipMemcpy(d_in2, in2, size, hipMemcpyHostToDevice);

	// set up CUDA timing
	// https://developer.nvidia.com/blog/how-implement-performance-metrics-cuda-cc/
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// execute kernels and time
	hipEventRecord(start);	
	add<<<numBlocks, *pBlockSize>>>(*pDataSize, d_in1, d_in2, d_out1);
	subtract<<<numBlocks, *pBlockSize>>>(*pDataSize, d_in1, d_in2, d_out1);
	multiply<<<numBlocks, *pBlockSize>>>(*pDataSize, d_in1, d_in2, d_out1);
	modulo<<<numBlocks, *pBlockSize>>>(*pDataSize, d_in1, d_in2, d_out1);
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(pTimer, start, stop);	
	
	// // print statement to verify accuracy of results
	// hipMemcpy(out1, d_out1, size, hipMemcpyDeviceToHost);
	// for (int j = 0; j < *pDataSize; j+=128) {
	// 	printf("out[%d] = %d\n", j, out1[j]);
	// }
		
	// clean up
	hipFree(d_in1); hipFree(d_in2); hipFree(d_out1);
	delete [] in1; 	delete [] in2; delete [] out1;
			
}

/* ========================================================================== */

int main(int argc, char** argv)
{
	using namespace std;
	
	// read command line arguments
	int totalThreads = 8192;
	int blockSize = 256;
	int dataSize = 8192;
	float timer = 0.0;
	int *pTotalThreads = &totalThreads;
	int *pBlockSize = &blockSize;
	int *pDataSize = &dataSize;
	float *pTimer = &timer;
	parse_cmdline(argc, argv, pTotalThreads, pBlockSize, pDataSize);
	totalThreads = *pTotalThreads;
	blockSize = *pBlockSize;
	dataSize = *pDataSize;

	// test harness
	int iterations = 10;
	float** res = new float*[3];
	for(int i = 0; i < 3; ++i) {
		res[i] = new float[iterations];
	}

	for (int i = 0; i < iterations; i++) {
		
		// constant memory test
		constant_test(pTotalThreads, pBlockSize, pDataSize, pTimer);
		res[0][i] = *pTimer;
		
		// shared memory test
		shared_test(pTotalThreads, pBlockSize, pDataSize, pTimer);
		res[1][i] = *pTimer;

		// global memory test
		global_test(pTotalThreads, pBlockSize, pDataSize, pTimer);
		res[2][i] = *pTimer;

	}

	// write results array to file
	FILE * pFile;
	pFile = fopen("results.txt","w");

	float sum = 0.0;
	for(int i = 0; i < iterations; i++) {
        sum += res[0][i];
		fprintf(pFile, "Constant Memory[%d] = %f\n", i, res[0][i]);
    }
	printf("Constant Memory Average = %f\n", (sum/iterations));
	fprintf(pFile, "Constant Memory Average = %f\n", (sum/iterations));
	
	sum = 0.0;	
	for(int i = 0; i < iterations; i++) {
        sum += res[1][i];
		fprintf(pFile, "Shared Memory[%d] = %f\n", i, res[1][i]);
    }
	printf("Shared Memory Average = %f\n", (sum/iterations));
	fprintf(pFile, "Shared Memory Average = %f\n", (sum/iterations));

	sum = 0.0;	
	for(int i = 0; i < iterations; i++) {
        sum += res[2][i];
		fprintf(pFile, "Global Memory[%d] = %f\n", i, res[2][i]);
    }
	printf("Global Memory Average = %f\n", (sum/iterations));
	fprintf(pFile, "Global Memory Average = %f\n", (sum/iterations));

	fclose(pFile);
	
}

/* ========================================================================== */

/* APPENDIX */

// hipError_t err = hipGetLastError();
    // if ( err != hipSuccess ) {
    // 	printf("CUDA Error: %s\n", hipGetErrorString(err)); 
    // 	// Possibly: exit(-1) if program cannot continue....
    // }
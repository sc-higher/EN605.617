/*
Sean Connor - March 2022
605.617 Module 8 Assignment
*/

/* ========================================================================== */

#include <stdio.h>
#include <iostream>
#include <numeric>
#include <iterator>
#include <chrono>
#include <random>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hipblas.h>
#include <hiprand/hiprand.h>

#define IDX2C(i,j,ld) (((j)*(ld))+(i))

static std::random_device rd;
static std::mt19937 rng{rd()};
static std::uniform_int_distribution<int> case2_val(0,3);
static std::uniform_int_distribution<int> case3_val(0,100);

/* ========================================================================== */

/**
 * @brief 
 * 
 * @param array Point to array to be filled.
 * @param len Length of array to be filled.
 * @param type Type 1 = Increment 0 - len; Type 2 = 0-3 random; Type 3 = random
 */
void in_generator(int * array, int len, int type) {

	switch(type) {
		case 1:
			for (int i = 0; i < len; i++) {
				array[i] = i;
			}
			break;
		case 2:
			for (int i = 0; i < len; i++) {
				array[i] = case2_val(rng);
			}
			break;
		case 3:
			for (int i = 0; i < len; i++) {
				array[i] = case3_val(rng);
			}
			break;
		default:
			std::cout << "INVALID" << std::endl;
	}
	
}

/* ========================================================================== */

/**
 * @brief 
 * 
 * @param argc 
 * @param argv 
 */
void parse_cmdline(int argc, char **argv, int * numTh, int * blSz, int * dataSz)
{
	if (argc >= 2) {
		*numTh = atoi(argv[1]);
	}
	if (argc >= 3) {
		*blSz = atoi(argv[2]);
	}
	if (argc >= 4) {
		*dataSz = atoi(argv[3]);
	}

	int numBlocks = *numTh / *blSz;	

	// validate command line arguments
	if (*numTh % *blSz != 0) {
		++numBlocks;
		*numTh = numBlocks * (*blSz);
		printf("Warning: Total thread count is not evenly divisible by the block size\n");
		printf("The total number of threads will be rounded up to %d\n", *numTh);
	}
	printf("================\n");
	printf("Total Threads: %d\n", *numTh);
	printf("Block Size: %d\n", *blSz);
	printf("Data Size: %d\n", *dataSz);
	printf("================\n");
}

/* ========================================================================== */

hipfftReal* generate_sine(int samples) {

	float pi = 3.14159;
	float amplitude = 1.0;
	float frequency = 10.0;
	float phase = 0.0;
	float sampleRate = 1000.0;

	hipfftReal *arr = new hipfftReal[samples];

	for (int i = 0; i < samples; i++)
  	{
		arr[i] = amplitude * sin(2.0 * pi * frequency * (i/sampleRate) + phase);
  	}

	return arr;

}

/* ========================================================================== */

void execute_fft_gpu(float * pTimer, int * pDataSize) {
	
	int NX = *pDataSize;

	// set up the cuFFT plan
	int batch = 1;
	hipfftHandle plan;
    hipfftPlan1d(&plan, NX, HIPFFT_R2C, batch);

	// copy the data to device
	hipfftReal *h_data = generate_sine(NX);
	hipComplex *r_data = (hipComplex*) malloc(sizeof(hipComplex)*NX);
	memset(r_data, 0, sizeof(hipComplex)*NX);

	// set up CUDA timing
	// https://developer.nvidia.com/blog/how-implement-performance-metrics-cuda-cc/
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// execute and time!
	hipEventRecord(start);
	
	hipfftReal *d_data_in;
	hipComplex *d_data_out;	
	hipMalloc((void **) &d_data_in, sizeof(hipfftReal)*NX);
	hipMalloc((void **) &d_data_out, sizeof(hipComplex)*NX);
	hipMemcpy(d_data_in, h_data, sizeof(hipfftReal)*NX, hipMemcpyHostToDevice);

	// execute the cuFFT
	hipfftExecR2C(plan, d_data_in, d_data_out);

	// copy results back to host
	hipMemcpy(r_data, d_data_out, sizeof(hipComplex)*NX, hipMemcpyDeviceToHost);

    // stop timer and record
    hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(pTimer, start, stop);	

	// // print statement to verify accuracy of results
	// float *res = new float[NX/2] {0.0};
	// for (int j = 1; j <= (NX/2); j++) {
	// 	res[j-1] = sqrt((r_data[j].x * r_data[j].x)+ (r_data[j].y * r_data[j].y));
	// 	if (res[j-1] > 5) {
	// 		printf("fft[%d] = %f\n", j-1, res[j-1]);
	// 	}
	// }

	// clean up
	hipfftDestroy(plan);
	hipFree(d_data_in); hipFree(d_data_out);
	free(r_data);
	delete [] h_data;

}

/* ========================================================================== */

void execute_cublas_gpu_test() {

    hipblasHandle_t handle;

	// int NX = *pDataSize;
	int M = 3;
	int N = 3;
	float * matrix = new float [M*N];

	for (int j = 0; j < N; j++) {
        for (int i = 0; i < M; i++) {
            matrix[IDX2C(i,j,M)] = 5.0;
        }
    }

	float *d_in1, *d_in2, *d_out1;
	hipMalloc((void **) &d_in1, sizeof(float)*M*N);
	hipMalloc((void **) &d_in2, sizeof(float)*M*N);
	hipMalloc((void **) &d_out1, sizeof(float)*M*N);
	hipblasCreate(&handle);
	hipblasSetMatrix (M, N, sizeof(float), matrix, M, d_in1, M);
	hipblasSetMatrix (M, N, sizeof(float), matrix, M, d_in2, M);
	cublasSgemm('n', 'n', M, N, M, 1, d_in1, M, d_in2, M, 0, d_out1, M);
	hipblasGetMatrix(M, N, sizeof(float), d_out1, M, matrix, M);

	for (int j = 0; j < N; j++) {
        for (int i = 0; i < M; i++) {
            printf("cublas[%d] = %f\n", i, matrix[IDX2C(i,j,M)]);
        }
    }

	// clean up
	hipFree (d_in1);
	hipFree (d_in2);
	hipFree (d_out1);
    // cublasDestroy_v2(handle);
	delete[] matrix;

}

/* ========================================================================== */

void execute_cublas_gpu(float * pTimer, int * pDataSize) {

    hipblasHandle_t handle;

	int NX = *pDataSize;
	int M, N = 0;
	M = sqrt(NX);
	N = M;
	int size = M*N;
	float * matrix = new float [size];
	

	// setup cuRAND and generate random matrix
	hiprandGenerator_t gen;
	hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_MTGP32);
	hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL);
	float * devRand;
	hipMalloc((void **) &devRand, sizeof(float)*M*N);	
	hiprandGenerateUniform(gen, devRand, size);
	hipMemcpy(matrix, devRand, sizeof(float)*size, hipMemcpyDeviceToHost);

	// // print matrix
	// for (int j = 0; j < N; j++) {
    //     for (int i = 0; i < M; i++) {
    //         // matrix[IDX2C(i,j,M)] = 5.0;
	// 		printf("cublas[%d] = %f\n", i, matrix[IDX2C(i,j,M)]);
    //     }
    // }

	// set up CUDA timing
	// https://developer.nvidia.com/blog/how-implement-performance-metrics-cuda-cc/
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// execute and time!
	hipEventRecord(start);

	float *d_in1, *d_in2, *d_out1;
	hipMalloc((void **) &d_in1, sizeof(float)*M*N);
	hipMalloc((void **) &d_in2, sizeof(float)*M*N);
	hipMalloc((void **) &d_out1, sizeof(float)*M*N);
	hipblasCreate(&handle);
	hipblasSetMatrix (M, N, sizeof(float), matrix, M, d_in1, M);
	hipblasSetMatrix (M, N, sizeof(float), matrix, M, d_in2, M);
	cublasSgemm('n', 'n', M, N, M, 1, d_in1, M, d_in2, M, 0, d_out1, M);
	hipblasGetMatrix(M, N, sizeof(float), d_out1, M, matrix, M);

    // stop timer and record
    hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(pTimer, start, stop);	

	// // print statement to verify accuracy of results
	// for (int j = 0; j < N; j++) {
    //     for (int i = 0; i < M; i++) {
    //         printf("cublas[%d] = %f\n", i, matrix[IDX2C(i,j,M)]);
    //     }
    // }

	// clean up
	hipFree (d_in1);
	hipFree (d_in2);
	hipFree (d_out1);
    // cublasDestroy_v2(handle);
	delete[] matrix;
	hiprandDestroyGenerator(gen);

}

/* ========================================================================== */

/**
 * @brief 
 * 
 * @param pTotalThreads 
 * @param pBlockSize 
 * @param pDataSize 
 * @param pTimer 
 */
void execute_gpu(int * pTotalThreads, int * pBlockSize, int * pDataSize, 
    float * pTimer) {

	// // test correctness of cuBLAS code
	// execute_cublas_gpu_test();

    // test harness
	int iterations = 10;
	float** res = new float*[2];
	for(int i = 0; i < 2; ++i) {
		res[i] = new float[iterations];
	}

	for (int i = 0; i < iterations; i++) {

		// CUFFT test
		execute_fft_gpu(pTimer, pDataSize);
		res[0][i] = *pTimer;

        // CUBLAS test
        execute_cublas_gpu(pTimer, pDataSize);
		res[1][i] = *pTimer;

	}

	// write results array to file
	FILE * pFile;
	pFile = fopen("results.txt","w");

	float sum = 0.0;
	for(int i = 1; i < iterations; i++) {
        sum += res[0][i];
		fprintf(pFile, "cuFFT[%d] = %f\n", i, res[0][i]);
    }
	printf("cuFFT Average = %f\n", (sum/iterations));
	fprintf(pFile, "cuFFT Average = %f\n", (sum/iterations));

    sum = 0.0;
	for(int i = 1; i < iterations; i++) {
        sum += res[1][i];
		fprintf(pFile, "cuBLAS[%d] = %f\n", i, res[1][i]);
    }
	printf("cuBLAS Average = %f\n", (sum/iterations));
	fprintf(pFile, "cuBLAS Average = %f\n", (sum/iterations));

	fclose(pFile);
    
}

/* ========================================================================== */

int main(int argc, char** argv) {
	
    using namespace std;
	
	// read command line arguments
	int totalThreads = 4096;
	int blockSize = 256;
	int dataSize = 4096;
	float timer = 0.0;
	int *pTotalThreads = &totalThreads;
	int *pBlockSize = &blockSize;
	int *pDataSize = &dataSize;
	float *pTimer = &timer;

	parse_cmdline(argc, argv, pTotalThreads, pBlockSize, pDataSize);
	execute_gpu(pTotalThreads, pBlockSize, pDataSize, pTimer);
	
}

/* ========================================================================== */

/* APPENDIX */

// CUDA ERROR CHECKING 

	// cudaError_t err = cudaGetLastError();
    // if ( err != cudaSuccess ) {
    // 	printf("CUDA Error: %s\n", cudaGetErrorString(err)); 
    // 	// Possibly: exit(-1) if program cannot continue....
    // }
/*
Sean Connor - May 2022
605.617 Final Project
*/

/* ========================================================================== */


#include <hip/hip_runtime.h>
#include <stdio.h>

#ifndef M_PI
    #define M_PI 3.14159265358979323846
#endif

// CFD Default Parameters
#define NX 101
#define NT 10001
#define ALPHA 0.1
#define T_MAX 5.0
#define LENGTH 1.0

// CUDA Default Parameters
#define TOTAL_THREADS 8192
#define BLOCK_SIZE 256

/* ========================================================================== */

/**
 * @brief 
 * 
 * @param data_size : length of data array
 * @param start : starting index for calculation
 * @param stop : ending index for calculation
 * @param r : ftcs equation constant
 * @param T : data array
 * @return __global__ 
 */
 __global__ 
 void ftcs(int data_size, int start, int stop, float r, float * T) {
	
	int nx = stop-start+3; // this is the length of each 'row'
    
    int i = threadIdx.x + start;

	T[i+nx] = r*T[i-1] + (1-2*r)*T[i] + r*T[i+1];
	
}

/* ========================================================================== */

/**
 * @brief 
 * 
 * @param total_threads 
 * @param block_size 
 * @param data_size 
 * @param nx number of space steps
 * @param nt number of time steps
 * @param dx size of space step
 * @param dt size of time step
 * @param alpha thermal diffusivity constant
 */
 void execute_ftcs(
    int total_threads,
    int block_size,
    int data_size, 
    int nx,
    int nt,
    float dx,
    float dt,
    float alpha,
    float * p_timer) {

	int size = data_size * sizeof(float);
	int num_blocks = total_threads / block_size;

	// allocate host data arrays 
	float *T = new float[data_size] {0.0};

    // set initial condition
    for (int i=1; i<nx-1; i++) {
        T[i] = 1.0;
    }

    // set boundary conditions
    float time = 0.0;
    for (int i=0; i<data_size; i += nx) {
        T[i] = 1 + sin(M_PI*time); //1.0;
        T[i + nx - 1] = 5.0;
        time += dt;
    }

    // set up CUDA timing
	// https://developer.nvidia.com/blog/how-implement-performance-metrics-cuda-cc/
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
    hipEventRecord(start);
	
	// allocate device and copy from host
	float *d_T;
	hipMalloc((void **) &d_T, size);
	hipMemcpy(d_T, T, size, hipMemcpyHostToDevice);	
	
    // all data kept in a single 1D array - need to index each time step
    int idx_start = 0;
    int idx_stop = 0;
    int width = nx-2;
    float r = alpha*dt/(dx*dx);

    // execute each time step in CUDA
    for (int i=0; i<nt; i++) {
        idx_start = i * nx + 1;
        idx_stop = idx_start + nx - 3;
        ftcs<<<1, width>>>(data_size, idx_start, idx_stop, r, d_T);
    }
	
	// copy data back to host
	hipMemcpy(T, d_T, size, hipMemcpyDeviceToHost);

    // stop CUDA timing
    hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(p_timer, start, stop);

    // make X and t linspaces for grid study
    float x_linspace[NX] = {0.0};
    float t_linspace[NT] = {0.0};
    for (int i=1; i<nx; i++) {
        x_linspace[i] = x_linspace[i-1] + dx;
    }
    for (int i=1; i<nt; i++) {
        t_linspace[i] = t_linspace[i-1] + dt;
    }

    // write results array to file (t,x,T) 
    // note gnuplot format requires a newline between 'blocks' for 3D data
    FILE * p_file;
    p_file = fopen("results.dat","w");
    int indx = 0;
    for (int i=0; i<nt; i++) {
        for (int j=0; j<nx; j++) {
            indx = i*nx + j;
            fprintf(p_file,"%.4f,%.2f,%.2f\n",t_linspace[i],x_linspace[j],T[indx]);
        }
        fprintf(p_file,"\n");
    }

    printf("Elapsed Time (ms): %.2f\n", *p_timer);

    fclose(p_file);
		
	// clean up
	hipFree(d_T);
	delete [] T;
			
}

/* ========================================================================== */

int main(int argc, char** argv)
{
	using namespace std;
	
	// CUDA Parameters
	int total_threads = TOTAL_THREADS;
	int block_size = BLOCK_SIZE;
    float timer = 0.0;
	float *p_timer = &timer;

    // CFD Parameters
    int nx = NX;
    int nt = NT;
    float alpha = ALPHA;
    float t_max = T_MAX;
    float length = LENGTH;
    int data_size = nx*nt;   
    float dx = length / (nx-1);
    float dt = t_max / (nt-1);

    // execute the FTCS kernel
	execute_ftcs(total_threads, block_size, data_size, nx, nt, dx, dt, alpha, p_timer);
    
}

/* ========================================================================== */

/* APPENDIX */

// cudaError_t err = cudaGetLastError();
    // if ( err != cudaSuccess ) {
    // 	printf("CUDA Error: %s\n", cudaGetErrorString(err)); 
    // 	// Possibly: exit(-1) if program cannot continue....
    // }
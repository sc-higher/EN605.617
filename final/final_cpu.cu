/*
Sean Connor - May 2022
605.617 Final Project
*/

/* ========================================================================== */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <chrono>

#ifndef M_PI
    #define M_PI 3.14159265358979323846
#endif

// CFD Default Parameters
#define NX 101
#define NT 10001
#define ALPHA 0.1
#define T_MAX 5.0
#define LENGTH 1.0

// CUDA Default Parameters
#define TOTAL_THREADS 8192
#define BLOCK_SIZE 256

/* ========================================================================== */

/**
 * @brief 
 * 
 * @param data_size : length of data array
 * @param start : starting index for calculation
 * @param stop : ending index for calculation
 * @param r : ftcs equation constant
 * @param T : data array
 * @return __global__ 
 */
 __global__ 
 void ftcs(int data_size, int start, int stop, float r, float * T) {
	
	for (int i = (blockIdx.x * blockDim.x) + threadIdx.x;
		 i < data_size;
		 i += blockDim.x * gridDim.x)
	{
		if ( (i>=start) && (i<=stop) ) {
            
            int nx = stop-start+3; // this is the length of each 'row'
            T[i+nx] = r*T[i-1] + (1-2*r)*T[i] + r*T[i+1];

        }
	}
	
}

/* ========================================================================== */

/**
 * @brief 
 * 
 * @param total_threads 
 * @param block_size 
 * @param data_size 
 * @param nx 
 * @param nt 
 * @param dx 
 * @param dt 
 * @param alpha 
 */
 void execute_ftcs(
    int total_threads,
    int block_size,
    int data_size, 
    int nx,
    int nt,
    float dx,
    float dt,
    float alpha,
    float * p_timer) {

	// allocate host data arrays 
	float *T = new float[data_size] {0.0};

    // set initial condition
    for (int i=1; i<nx-1; i++) {
        T[i] = 1.0;
    }

    // set boundary conditions
    float time = 0.0;
    for (int i=0; i<data_size; i += nx) {
        T[i] = 1 + sin(M_PI*time); //1.0;
        T[i + nx - 1] = 5.0;
        time += dt;
    }

    // start timer
    auto start = std::chrono::high_resolution_clock::now();
	
    // all data kept in a single 1D array - need to index each time step
    int idx_start = 0;
    int idx_stop = 0;
    float r = alpha*dt/(dx*dx);

    // execute each time step in CPU
    for (int i=0; i<nt; i++) {
        idx_start = i * nx + 1;
        idx_stop = idx_start + nx - 3;

        for (int j=idx_start; j<=idx_stop; j++) {
            T[j+nx] = r*T[j-1] + (1-2*r)*T[j] + r*T[j+1];
        }
        
    }

    // stop timer
    auto stop = std::chrono::high_resolution_clock::now();
	auto duration = std::chrono::duration_cast<std::chrono::microseconds>(stop - start);

    printf("Duration: %.2fms\n", (float)duration.count()/1000);

    // make X and t linspaces for grid study
    float x_linspace[NX] = {0.0};
    float t_linspace[NT] = {0.0};
    for (int i=1; i<nx; i++) {
        x_linspace[i] = x_linspace[i-1] + dx;
    }
    for (int i=1; i<nt; i++) {
        t_linspace[i] = t_linspace[i-1] + dt;
    }

    // write results array to file (t,x,T) 
    // note gnuplot format requires a newline between 'blocks' for 3D data
    FILE * p_file;
    p_file = fopen("results.dat","w");
    int indx = 0;
    for (int i=0; i<nt; i++) {
        for (int j=0; j<nx; j++) {
            indx = i*nx + j;
            fprintf(p_file,"%.4f,%.2f,%.2f\n",t_linspace[i],x_linspace[j],T[indx]);
        }
        fprintf(p_file,"\n");
    }
    
    // Alternate format matrix-style
    // for (int i=0; i<data_size; i++) {
    //     if (i % *p_nx == 0) {
    //         fprintf(p_file, "\n");
    //     }
    //     fprintf(p_file,"%.2f ",T[i]);
    // }

    fclose(p_file);
		
	// clean up
	delete [] T;
			
}

/* ========================================================================== */

int main(int argc, char** argv)
{
	using namespace std;
	
	// CUDA Parameters
	int total_threads = TOTAL_THREADS;
	int block_size = BLOCK_SIZE;
    float timer = 0.0;
	float *p_timer = &timer;

    // CFD Parameters
    int nx = NX;
    int nt = NT;
    float alpha = ALPHA;
    float t_max = T_MAX;
    float length = LENGTH;
    int data_size = nx*nt;   
    float dx = length / (nx-1);
    float dt = t_max / (nt-1);

	execute_ftcs(total_threads, block_size, data_size, nx, nt, dx, dt, alpha, p_timer);
    
}

/* ========================================================================== */

/* APPENDIX */

// cudaError_t err = cudaGetLastError();
    // if ( err != cudaSuccess ) {
    // 	printf("CUDA Error: %s\n", cudaGetErrorString(err)); 
    // 	// Possibly: exit(-1) if program cannot continue....
    // }
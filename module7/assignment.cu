/*
Sean Connor - March 2022
605.617 Module 7 Assignment
*/

/* ========================================================================== */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <numeric>
#include <iterator>
#include <chrono>
#include <random>

static std::random_device rd;
static std::mt19937 rng{rd()};
static std::uniform_int_distribution<int> case2_val(0,3);
static std::uniform_int_distribution<int> case3_val(0,100);

/* ========================================================================== */

/**
 * @brief 
 * 
 * @param n 
 * @param a 
 * @param b 
 * @param c 
 * @return __global__ 
 */
 __global__ 
 void add(int n, int *a, int *b, int *c) {
	
	for (int i = (blockIdx.x * blockDim.x) + threadIdx.x;
		 i < n;
		 i += blockDim.x * gridDim.x)
	{
		c[i] = a[i] + b[i];
	}
	
}

/* ========================================================================== */

/**
 * @brief 
 * 
 * @param n 
 * @param a 
 * @param b 
 * @param c 
 * @return __global__ 
 */
__global__ 
void subtract(int n, int *a, int *b, int *c) {
	
	for (int i = (blockIdx.x * blockDim.x) + threadIdx.x;
		 i < n;
		 i += blockDim.x * gridDim.x)
	{
		c[i] = a[i] - b[i];
	}
	
}

/* ========================================================================== */

/**
 * @brief 
 * 
 * @param n 
 * @param a 
 * @param b 
 * @param c 
 * @return __global__ 
 */
__global__ 
void multiply(int n, int *a, int *b, int *c) {
	
	for (int i = (blockIdx.x * blockDim.x) + threadIdx.x;
		 i < n;
		 i += blockDim.x * gridDim.x)
	{
		c[i] = a[i] * b[i];
	}
	
}

/* ========================================================================== */

/**
 * @brief 
 * 
 * @param n 
 * @param a 
 * @param b 
 * @param c 
 * @return __global__ 
 */
__global__ 
void modulo(int n, int *a, int *b, int *c) {
	
	for (int i = (blockIdx.x * blockDim.x) + threadIdx.x;
		 i < n;
		 i += blockDim.x * gridDim.x)
	{
		c[i] = a[i] % b[i];
	}
	
}

/* ========================================================================== */

/**
 * @brief 
 * 
 * @param array Point to array to be filled.
 * @param len Length of array to be filled.
 * @param type Type 1 = Increment 0 - len; Type 2 = 0-3 random; Type 3 = random
 */
void in_generator(int * array, int len, int type) {

	switch(type) {
		case 1:
			for (int i = 0; i < len; i++) {
				array[i] = i;
			}
			break;
		case 2:
			for (int i = 0; i < len; i++) {
				array[i] = case2_val(rng);
			}
			break;
		case 3:
			for (int i = 0; i < len; i++) {
				array[i] = case3_val(rng);
			}
			break;
		default:
			std::cout << "INVALID" << std::endl;
	}
	
}

/* ========================================================================== */

/**
 * @brief 
 * 
 * @param argc 
 * @param argv 
 */
void parse_cmdline(int argc, char **argv, int * numTh, int * blSz, int * dataSz)
{
	if (argc >= 2) {
		*numTh = atoi(argv[1]);
	}
	if (argc >= 3) {
		*blSz = atoi(argv[2]);
	}
	if (argc >= 4) {
		*dataSz = atoi(argv[3]);
	}

	int numBlocks = *numTh / *blSz;	

	// validate command line arguments
	if (*numTh % *blSz != 0) {
		++numBlocks;
		*numTh = numBlocks * (*blSz);
		printf("Warning: Total thread count is not evenly divisible by the block size\n");
		printf("The total number of threads will be rounded up to %d\n", *numTh);
	}
	printf("================\n");
	printf("Total Threads: %d\n", *numTh);
	printf("Block Size: %d\n", *blSz);
	printf("Data Size: %d\n", *dataSz);
	printf("================\n");
}

/* ========================================================================== */

/**
 * @brief 
 * 
 * @param pTotalThreads 
 * @param pBlockSize 
 * @param pDataSize 
 * @param pTimer 
 */
 void global_test(int * pTotalThreads, int * pBlockSize, int * pDataSize, 
	float * pTimer) {

	int size = *pDataSize * sizeof(int);
	int numBlocks = *pTotalThreads / *pBlockSize;

	// allocate host data arrays
	int *in1 = new int[*pDataSize] {0};
	int *in2 = new int[*pDataSize] {0};
	int *out1 = new int[*pDataSize] {0};
    int *out2 = new int[*pDataSize] {0};
    int *out3 = new int[*pDataSize] {0};
    int *out4 = new int[*pDataSize] {0};
	
	// generate data and allocate device data (global)
	int *d_in1, *d_in2, *d_out1, *d_out2, *d_out3, *d_out4;
	in_generator(in1,*pDataSize,1);
	in_generator(in2,*pDataSize,2);
	hipMalloc((void **) &d_in1, size);
	hipMalloc((void **) &d_in2, size);
	hipMalloc((void **) &d_out1, size);
    hipMalloc((void **) &d_out2, size);
    hipMalloc((void **) &d_out3, size);
    hipMalloc((void **) &d_out4, size);

	// set up CUDA timing
	// https://developer.nvidia.com/blog/how-implement-performance-metrics-cuda-cc/
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// execute and time!
	hipEventRecord(start);	

    // memcpy to device
    hipMemcpy(d_in1, in1, size, hipMemcpyHostToDevice);
	hipMemcpy(d_in2, in2, size, hipMemcpyHostToDevice);

    // execute kernels
	add<<<numBlocks, *pBlockSize>>>(*pDataSize, d_in1, d_in2, d_out1);
	subtract<<<numBlocks, *pBlockSize>>>(*pDataSize, d_in1, d_in2, d_out2);
	multiply<<<numBlocks, *pBlockSize>>>(*pDataSize, d_in1, d_in2, d_out3);
	modulo<<<numBlocks, *pBlockSize>>>(*pDataSize, d_in1, d_in2, d_out4);

    // memcpy again
    hipMemcpy(out1, d_out1, size, hipMemcpyDeviceToHost);
    hipMemcpy(out2, d_out2, size, hipMemcpyDeviceToHost);
    hipMemcpy(out3, d_out3, size, hipMemcpyDeviceToHost);
    hipMemcpy(out4, d_out4, size, hipMemcpyDeviceToHost);

    // stop timer and record
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(pTimer, start, stop);	
	
	// print statement to verify accuracy of results
	// for (int j = 0; j < *pDataSize; j+=512) {
	// 	printf("add[%d] = %d\n", j, out1[j]);
    //     printf("sub[%d] = %d\n", j, out2[j]);
    //     printf("mul[%d] = %d\n", j, out3[j]);
    //     printf("mod[%d] = %d\n", j, out4[j]);
	// }
		
	// clean up
	hipFree(d_in1); hipFree(d_in2); hipFree(d_out1);
    hipFree(d_out2); hipFree(d_out3); hipFree(d_out4);
	delete [] in1; 	delete [] in2; delete [] out1;
    delete [] out2; delete [] out3; delete [] out4;
			
}

/* ========================================================================== */

void stream_test(int * pTotalThreads, int * pBlockSize, int * pDataSize, 
    float * pTimer) {
	
	int size = *pDataSize * sizeof(int);
	int numBlocks = *pTotalThreads / *pBlockSize;

	// pinned memory -- host
	int *h_in1, *h_in2, *h_out1, *h_out2, *h_out3, *h_out4;
    hipHostMalloc((void **) &h_in1, size, hipHostMallocDefault);
    hipHostMalloc((void **) &h_in2, size, hipHostMallocDefault);
	hipHostMalloc((void **) &h_out1, size, hipHostMallocDefault);
    hipHostMalloc((void **) &h_out2, size, hipHostMallocDefault);
    hipHostMalloc((void **) &h_out3, size, hipHostMallocDefault);
    hipHostMalloc((void **) &h_out4, size, hipHostMallocDefault);
	in_generator(h_in1,*pDataSize,1);
	in_generator(h_in2,*pDataSize,2);
	memset(h_out1, 0, size);
    memset(h_out2, 0, size);
    memset(h_out3, 0, size);
    memset(h_out4, 0, size);	

	// pinned memory -- device
	int *d_in1, *d_in2, *d_out1, *d_out2, *d_out3, *d_out4;
	hipMalloc((void **) &d_in1, size);
	hipMalloc((void **) &d_in2, size);
	hipMalloc((void **) &d_out1, size);
    hipMalloc((void **) &d_out2, size);
    hipMalloc((void **) &d_out3, size);
    hipMalloc((void **) &d_out4, size);

    // set up CUDA timing
	// https://developer.nvidia.com/blog/how-implement-performance-metrics-cuda-cc/
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
    
    // stream and event setup 
    hipStream_t stream1, stream2, stream3, stream4;
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);
    hipStreamCreate(&stream3);
    hipStreamCreate(&stream4);

    hipEvent_t event1, event2;
    hipEventCreate(&event1);
    hipEventCreate(&event2);

    // execute and time!
	hipEventRecord(start);

    // memcpy to device -- sync at end before starting kernel exec
	hipMemcpyAsync(d_in1, h_in1, size, hipMemcpyHostToDevice, stream1);
	hipMemcpyAsync(d_in2, h_in2, size, hipMemcpyHostToDevice, stream2);
    hipEventRecord(event1);
    hipEventRecord(event2);

    // execute kernels
	add<<<numBlocks,*pBlockSize,0,stream1>>>(*pDataSize, d_in1, d_in2, d_out1);
    subtract<<<numBlocks,*pBlockSize,0,stream2>>>(*pDataSize, d_in1, d_in2, d_out2);
    multiply<<<numBlocks,*pBlockSize,0,stream3>>>(*pDataSize, d_in1, d_in2, d_out3);
    modulo<<<numBlocks,*pBlockSize,0,stream4>>>(*pDataSize, d_in1, d_in2, d_out4);

    // memcpy again
    hipMemcpyAsync(h_out1, d_out1, size, hipMemcpyDeviceToHost, stream1);
    hipMemcpyAsync(h_out2, d_out2, size, hipMemcpyDeviceToHost, stream2);
    hipMemcpyAsync(h_out3, d_out3, size, hipMemcpyDeviceToHost, stream3);
    hipMemcpyAsync(h_out4, d_out4, size, hipMemcpyDeviceToHost, stream4);

    // stop timer and record
    hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(pTimer, start, stop);
    
    // // print statement to verify accuracy of results
	// for (int j = 0; j < *pDataSize; j+=512) {
	// 	printf("add[%d] = %d\n", j, h_out1[j]);
    //     printf("sub[%d] = %d\n", j, h_out2[j]);
    //     printf("mul[%d] = %d\n", j, h_out3[j]);
    //     printf("mod[%d] = %d\n", j, h_out4[j]);
	// }
	
	// clean up
	hipHostFree(h_in1); hipHostFree(h_in2); hipHostFree(h_out1); 
    hipHostFree(h_out2); hipHostFree(h_out3); hipHostFree(h_out4);
	hipFree(d_in1); hipFree(d_in2); hipFree(d_out1);
    hipFree(d_out2); hipFree(d_out3); hipFree(d_out4);
    hipStreamDestroy(stream1); hipStreamDestroy(stream2);
    hipStreamDestroy(stream3); hipStreamDestroy(stream4);

}

/* ========================================================================== */

/**
 * @brief 
 * 
 * @param pTotalThreads 
 * @param pBlockSize 
 * @param pDataSize 
 * @param pTimer 
 */
void execute_gpu(int * pTotalThreads, int * pBlockSize, int * pDataSize, 
    float * pTimer) {

    // test harness
	int iterations = 10;
	float** res = new float*[2];
	for(int i = 0; i < 2; ++i) {
		res[i] = new float[iterations];
	}

	for (int i = 0; i < iterations; i++) {

		// global memory test
		global_test(pTotalThreads, pBlockSize, pDataSize, pTimer);
		res[0][i] = *pTimer;

        // stream test
        stream_test(pTotalThreads, pBlockSize, pDataSize, pTimer);
		res[1][i] = *pTimer;

	}

	// write results array to file
	FILE * pFile;
	pFile = fopen("results.txt","w");

	float sum = 0.0;
	for(int i = 0; i < iterations; i++) {
        sum += res[0][i];
		fprintf(pFile, "Global Memory[%d] = %f\n", i, res[0][i]);
    }
	printf("Global Memory Average = %f\n", (sum/iterations));
	fprintf(pFile, "Global Memory Average = %f\n", (sum/iterations));

    sum = 0.0;
	for(int i = 0; i < iterations; i++) {
        sum += res[1][i];
		fprintf(pFile, "Stream[%d] = %f\n", i, res[1][i]);
    }
	printf("Stream Average = %f\n", (sum/iterations));
	fprintf(pFile, "Stream Average = %f\n", (sum/iterations));

	fclose(pFile);
    
}

/* ========================================================================== */

int main(int argc, char** argv) {
	
    using namespace std;
	
	// read command line arguments
	int totalThreads = 4096;
	int blockSize = 256;
	int dataSize = 4096;
	float timer = 0.0;
	int *pTotalThreads = &totalThreads;
	int *pBlockSize = &blockSize;
	int *pDataSize = &dataSize;
	float *pTimer = &timer;

	parse_cmdline(argc, argv, pTotalThreads, pBlockSize, pDataSize);
    execute_gpu(pTotalThreads, pBlockSize, pDataSize, pTimer);
	
}

/* ========================================================================== */

/* APPENDIX */

// cudaError_t err = cudaGetLastError();
    // if ( err != cudaSuccess ) {
    // 	printf("CUDA Error: %s\n", cudaGetErrorString(err)); 
    // 	// Possibly: exit(-1) if program cannot continue....
    // }